//MULTIPLICACIÓN DE MATRICES(APLANADAS)NO CUADRADAS EN C++ y CUDA con tiempo
#include<iostream>
#include<stdio.h>
#include<malloc.h>
#include<hip/hip_runtime.h>
using namespace std; 


__global__ 
void MultiplicaMatricesCU(int* A,int filA,int colA,int* B,int filB,int colB,int* C){//filC=filA,colC=colB
	int row = blockIdx.y*blockDim.y + threadIdx.y;
	int col = blockIdx.x*blockDim.x + threadIdx.x;
	if((row<filA)&&(col<colB)){
		int suma=0;
		for(int k=0;k<filB;k++){//Se mueve entre las filas de B 
			suma=suma+A[(row*colA)+k]*B[(k*colB)+col];
		}
		C[(row*colB)+col]=suma;
	}	
}

__host__ 
void multiplicaMatrices(int* X,int filX,int colX,int* Y,int filY,int colY,int* Z){
	for(int i=0;i<filX;i++){
		for(int j=0;j<colY;j++){
			int suma=0;
			for(int k=0;k<filY;k++){
				suma=suma+X[(i*colX)+k]*Y[(k*colY)+j];

			}
			Z[(i*colY)+j]=suma;
		}	
	}
}

__host__ 
void imprime(int* A,int filas, int columnas){//imprime como si fuera una matriz
	for(int i = 0; i < filas; i++){
        	for(int j = 0; j < columnas; j++){
            		cout<<A[(i*columnas)+j]<<" ";
        	}
        cout<<endl;
    }
}	

__host__ 
bool compara(int *A, int *B, int filas, int columnas){
	for(int i = 0; i < filas; i++){
		for(int j = 0; j < columnas; j++){
			if(A[i*columnas+j] != B[i*columnas+j]) return false;
		}
	}
	return true;
}

__host__ 
void inicializa(int *A,int filas, int columnas){//inicializa arreglos
	for(int i=0;i<filas*columnas;i++){
		A[i]=1;
	}
}

int main(void){

	clock_t startCPU,endCPU,startGPU,endGPU;  
        hipError_t error = hipSuccess;
	int *A,*B,*C; //A[filA][colA],B[filB][colB],C[filA][colB]
	int *d_A,*d_B,*d_C,*h_C;
	int filA=1024,colA=1024,filB=1024,colB=1024;
	//int filA=5,colA=10,filB=10,colB=1;
	//-------------------------------CPU--------------------------------------------------------------------
	startCPU = clock();	

	A=(int*)malloc(filA*colA*sizeof(int)); 
	B=(int*)malloc(filB*colB*sizeof(int));
	C=(int*)malloc(filA*colB*sizeof(int));

	inicializa(A,filA,colA);
	inicializa(B,filB,colB);
	
	if(colA==filB){//para que sean multiplicables
		multiplicaMatrices(A,filA,colA,B,filB,colB,C);
		//imprime(C,filA,colB);
	}else{
		cout<<"Error, no se pueden multiplicar"<<endl;
		return 0;
	}
	
	endCPU = clock();

	double time_CPU=((double)(endCPU-startCPU))/CLOCKS_PER_SEC;
	cout<<"El tiempo transcurrido en la CPU fue: "<<time_CPU<<endl;
	//-------------------------------GPU--------------------------------------------------------------------
	h_C=(int*)malloc(filA*colB*sizeof(int));

	startGPU = clock();

	error=hipMalloc((void**)&d_A,filA*colA*sizeof(int));
        if(error != hipSuccess){
            cout<<"Error reservando memoria para d_A"<<endl;
            //return -1;
        }
    
	hipMalloc((void**)&d_B,filB*colB*sizeof(int));
        if(error != hipSuccess){
            cout<<"Error reservando memoria para d_B"<<endl;
            //return -1;
        }
        
	hipMalloc((void**)&d_C,filA*colB*sizeof(int));	
        if(error != hipSuccess){
            cout<<"Error reservando memoria para d_C"<<endl;
            //return -1;
        }
	
	hipMemcpy(d_A,A,filA*colA*sizeof(int),hipMemcpyHostToDevice);//destino d_A y origen A
	hipMemcpy(d_B,B,filB*colB*sizeof(int),hipMemcpyHostToDevice);	

	//Depende directamente de la dimensión de las matrices
	dim3 dimblock(32,32,1);
	dim3 dimGrid(ceil((double)(colB/32)),ceil((double)(filA/32)),1);
	
	MultiplicaMatricesCU<<<dimGrid,dimblock>>>(d_A,filA,colA,d_B,filB,colB,d_C);

	hipDeviceSynchronize();

	hipMemcpy(h_C,d_C,filA*colB*sizeof(int),hipMemcpyDeviceToHost);
	
	endGPU = clock();

	//imprime(h_C,filA,colB);
	double time_GPU=((double)(endGPU-startGPU))/CLOCKS_PER_SEC;
	cout<<"El tiempo transcurrido en la GPU fue: "<<time_GPU<<endl;
	//-----------------------------------------------------------------------------------
	cout<<"El tiempo de aceleramiento fue: "<<time_CPU/time_GPU<<endl;
		
	if(compara(h_C, C, filA, colB)) cout << "Buen cálculo" << endl;
	else cout << "Mal cálculo" << endl;
	
	free(A);free(B);free(C);free(h_C);
	hipFree(d_A);
	hipFree(d_B);
	hipFree(d_C);
	return 0;
}